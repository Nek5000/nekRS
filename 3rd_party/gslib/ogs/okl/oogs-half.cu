#include "hip/hip_runtime.h"
#include <hip/hip_fp16.h>

extern "C" __global__ void packBuf_halfAdd(
  const int N,
  const int Nentries,
  const int stride,
  const int * __restrict__ gatherStarts,
  const int * __restrict__ gatherIds,
  const int * __restrict__ scatterStarts,
  const int * __restrict__ scatterIds,
  float * __restrict__ q,
  half * __restrict__ qout)
{
  const int id = blockDim.x * blockIdx.x + threadIdx.x;
  if (id < N * Nentries) {
    const int sid = id % N;
    const int k = id / N;
    const int startGather = gatherStarts[sid];
    const int endGather = gatherStarts[sid + 1];
    const int startScatter = scatterStarts[sid];
    const int endScatter= scatterStarts[sid + 1];

    float gq = 0.0f;
    for(dlong n=startGather;n<endGather;++n){
      const dlong id = gatherIds[n];
      gq += q[id+k*stride];
    }
    for(dlong n=startGather;n<endGather;++n){
      const dlong id = gatherIds[n];
      q[id+k*stride] = gq;
    }

    for(dlong n=startScatter;n<endScatter;++n){
      const dlong id = scatterIds[n];
      qout[id*Nentries+k] = __float2half(gq);
    }
  }
}

extern "C" __global__ void unpackBuf_halfAdd(
  const int N,
  const int Nentries,
  const int stride,
  const int * __restrict__ gatherStarts,
  const int * __restrict__ gatherIds,
  const int * __restrict__ scatterStarts,
  const int * __restrict__ scatterIds,
  const half * __restrict__ q,
  float * __restrict__ qout) 
{
  const int id = blockDim.x * blockIdx.x + threadIdx.x;
  if (id < N * Nentries) {
    const int gid = id % N;
    const int k = id / N;
    const dlong startGather = gatherStarts[gid];
    const dlong endGather = gatherStarts[gid+1];
    const dlong startScatter = scatterStarts[gid];
    const dlong endScatter = scatterStarts[gid+1];

    float gq = 0.0f;
    for(dlong n=startGather;n<endGather;++n){
      const dlong id = gatherIds[n];
      gq += __half2float(q[id*Nentries+k]);
    }
    
    for(dlong n=startScatter;n<endScatter;++n){
      const dlong id = scatterIds[n];
      qout[id+k*stride] += gq;
    }
  }
}
